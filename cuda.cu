#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

// Texture reference for reading image
static texture<guchar, 2> tex;

// arrays are optimized for 2D access so we'll use arrays
// insted of single row pointer memory addresses
static hipArray *array = NULL;

// Shared Mem on the dev is declared with __shared__
extern __shared__ unsigned char LocalBlock[];


void filter(
		guchar* d_image, gint width, gint height,
		guint channels, cuda_filter mode,
		gint radius, gint offset) {

	dim3 blockDim( 16, 16, 1);
	dim3 gridDim( width / blockDim.x + 1, height / blockDim.y + 1, 1);
	guint step = channels * width;

	switch ( mode) {
		case GREY:
			greyGRAY<<< gridDim, blockDim, 0 >>>( d_image, width, height, channels, step);
			break;
			
		case BOX:
			box<<< gridDim, blockDim, 0 >>>( d_image, width, height, channels, step);
			break;
			
		case SOBEL:
			break;
			
		case AVERAGE:
			break;
			
		default:
			g_printerr("Filter not found");
			break;
	}

}

extern "C" void cuda_init( ) {
}

extern "C" void setupTexture(gint width, gint height) {
	hipChannelFormatDesc desc;

// 	desc = hipCreateChannelDesc<unsigned char>();
	int e = (int)sizeof(guchar) * 8;
	desc = hipCreateChannelDesc(e, 0, 0, 0, hipChannelFormatKindUnsigned);

	cutilSafeCall(hipMallocArray(&array, &desc, width, height));
}

extern "C" void updateTexture(gint width, gint height, guchar *data, gint channel) {
	cutilSafeCall(hipMemcpyToArray(
			array,
			0, 0,
			data, /* 0 0 <- h und w offset */
			channel * sizeof(guchar) * width * height, hipMemcpyHostToDevice));
}

extern "C" void deleteTexture( ) {
	cutilSafeCall(hipFreeArray(array));
}

extern "C" void bindTexture( ) {
	/* clamp x and y axis to the boarder */
	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.filterMode = hipFilterModePoint;
	tex.normalized = false;

	cutilSafeCall(hipBindTextureToArray(tex, array));
}

extern "C" void unbindTexture( ) {
	cutilSafeCall(hipUnbindTexture(tex));
}
