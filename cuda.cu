#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

// Texture reference for reading image
texture<guchar, 2> tex;

// arrays are optimized for 2D access so we'll use arrays
// insted of single row pointer memory addresses
hipArray *array = NULL;

// Shared Mem on the dev is declared with __shared__
extern __shared__ unsigned char LocalBlock[];


void filter(
		guchar* d_image, gint width, gint height,
		guint channels) {

	guint step = channels * width;
	int x = 0, y = 0;

	switch ( filterParm.cuda_filter) {
		case GREY:
			grey<<< height, 384, 0 >>>( d_image, width, height, channels, step, filterParm);
			break;

		case BOX:
			box<<< height, 384, 0 >>>( d_image, width, height, channels, step, filterParm);
			break;

		case SOBEL:
			sobelTex<<< height, 384, 0 >>>( d_image, width, height, channels, step, filterParm);
			break;

		case AVERAGE:
			break;

		default:
			g_printerr("Filter not found");
			break;
	}

}

extern "C" void cuda_init( ) {
}

extern "C" void setupTexture( gint width, gint height) {
	hipChannelFormatDesc desc;

// 	desc = hipCreateChannelDesc<unsigned char>();
	int e = (int)sizeof( guchar) * 8;
	desc = hipCreateChannelDesc(e, 0, 0, 0, hipChannelFormatKindUnsigned);

	cutilSafeCall(hipMallocArray( &array, &desc, width, height));
}

extern "C" void bindTexture( ) {
	/* clamp x and y axis to the boarder */
	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.filterMode = hipFilterModePoint;
	tex.normalized = false;

	cutilSafeCall( hipBindTextureToArray( tex, array));
}

extern "C" void updateTexture( gint width, gint height, guchar *data, gint channel) {
	cutilSafeCall(hipMemcpyToArray(
			array,
			0, 0, /* 0 0 <- h und w offset */
			data,
			channel * sizeof( guchar) * width * height, hipMemcpyHostToDevice));
}

extern "C" void unbindTexture( ) {
	cutilSafeCall( hipUnbindTexture( tex));
}

extern "C" void deleteTexture( ) {
	cutilSafeCall( hipFreeArray( array));
}


/*
 * ALL the CUDA Functions
 */

__device__ unsigned char
ComputeSobel(unsigned char ul, // upper left
        unsigned char um, // upper middle
        unsigned char ur, // upper right
        unsigned char ml, // middle left
        unsigned char mm, // middle (unused)
        unsigned char mr, // middle right
        unsigned char ll, // lower left
        unsigned char lm, // lower middle
        unsigned char lr, // lower right
        float fScale )
{
    short Horz = ul + 2*ml + ll - ur - 2*mr - lr;
    short Vert = ul + 2*um + ur - ll - 2*lm - lr;

    short Sum = (short) (fScale*(abs(Horz)+abs(Vert)));
    if ( Sum < 0 ) return 0; else if ( Sum > 0xff ) return 0xff;
    return (unsigned char) Sum;
}

__global__ void sobelTex( guchar *d_image, gint width, gint height, guint channels, guint step, FilterParameter filterParm) {

	double fScale = 1.0;

    unsigned char *pSobel =
        (unsigned char *) (((char *) d_image)+blockIdx.x*step);
    for ( int i = threadIdx.x; i < width; i += blockDim.x ) {
        unsigned char pix00 = tex2D( tex, (float) i-1, (float) blockIdx.x-1 );
        unsigned char pix01 = tex2D( tex, (float) i+0, (float) blockIdx.x-1 );
        unsigned char pix02 = tex2D( tex, (float) i+1, (float) blockIdx.x-1 );
        unsigned char pix10 = tex2D( tex, (float) i-1, (float) blockIdx.x+0 );
        unsigned char pix11 = tex2D( tex, (float) i+0, (float) blockIdx.x+0 );
        unsigned char pix12 = tex2D( tex, (float) i+1, (float) blockIdx.x+0 );
        unsigned char pix20 = tex2D( tex, (float) i-1, (float) blockIdx.x+1 );
        unsigned char pix21 = tex2D( tex, (float) i+0, (float) blockIdx.x+1 );
        unsigned char pix22 = tex2D( tex, (float) i+1, (float) blockIdx.x+1 );
        pSobel[i] = ComputeSobel(pix00, pix01, pix02,
                pix10, pix11, pix12,
                pix20, pix21, pix22, fScale );
    }

}

__global__ void box( guchar *d_image, gint width, gint height, guint channels, guint step, FilterParameter filterParm) {

    // blockIdx.x * Pitch (image.width) = Startpointer auf die Idx.x te Zeile
    unsigned char *p =
        (unsigned char *) (((char *) d_image)+blockIdx.x*step);
    int b = 0;

    for ( int i = threadIdx.x; i < width; i += blockDim.x ) {
        if(b==1) {
            if(blockIdx.x % 2 ) {
                p[i] = 255;
            } else {
                p[i] = 0;
            }
        } else {
            if(blockIdx.x % 2 ) {
                p[i] = 0;
            } else {
                p[i] = 255;
            }
        }
        b=1;
    }

}

/* OLD OUTDATED CODE */
// __global__ void greyRGB( guchar* d_image, gint width, gint height, guint channels, guint step, FilterParameter filterParm) {
// 
// 	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
// 	x *= channels;
// 	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
// 	d_image[y*step+x]   = 128;
// 	d_image[y*step+x+1] = 128;
// 	d_image[y*step+x+2] = 128;
// 
// }

__global__ void grey( guchar* d_image, gint width, gint height, guint channels, guint step, FilterParameter filterParm) {

	for ( int i = threadIdx.x; i < width; i += blockDim.x ) {
		d_image[blockIdx.x*step+i] = filterParm.radius;
	}

}

__global__ void
sobelSharedTex(
		uchar4 *pSobelOriginal, unsigned short SobelPitch,
		short BlockWidth, short SharedPitch,
		short w, short h, float fScale, FilterParameter filterParm )
{

	int Radius = filterParm.radius;
    // pSobelOriginal > Pointer auf den Speicher in der Graka
    // SobelPitch > 1280 (BilderBreite)
    // BlockWidth > 80
    // SharedPitch > 384 // Zeilenlaenge
    // w > 1280
    // h > 1024
    // sharedMem > 2304 >> 48 * 48 = 2304
    // threads > 16,4
    // block   > 4,256
    // Radius 1 = Ich brauche links/rechts/oben/unten 1 extra pixel
    // radius darf max die haelfte des Blockes sein in x und y

    // u und v sind die KOs des Pixels, das ich kopieren will
    // u ist 4*80 = 320 -> 4*320 = 1280  -->  Der 320er Anfang jedes Blockes
    // auf u (anfang des 320er Blockes) muss dann noch der Zu nehmende Pixel addiert werden
    short u = 4*blockIdx.x*BlockWidth;
    short v = blockIdx.y*blockDim.y + threadIdx.y;
    short ib;

    // SharedIdx > Zeilenanfang vom SharedMem
    // 384 > Zeilenbreite vom SharedMem
    int SharedIdx = threadIdx.y * SharedPitch;

    // ib geht komplett durch von 0-81
    // ib geht 16er schritte
    // 4*ib = 64
    // damit hat man einheitliches lesen
    // t0 liest 4byte
    // t1 liest 4byte
    // -> 16Threads a 4byte = 64byte
    for ( ib = threadIdx.x; ib < BlockWidth+2*Radius; ib += blockDim.x ) {
        LocalBlock[SharedIdx+4*ib+0] = tex2D( tex,
                (float) (u+4*ib-Radius+0), (float) (v-Radius) );
        LocalBlock[SharedIdx+4*ib+1] = tex2D( tex,
                (float) (u+4*ib-Radius+1), (float) (v-Radius) );
        LocalBlock[SharedIdx+4*ib+2] = tex2D( tex,
                (float) (u+4*ib-Radius+2), (float) (v-Radius) );
        LocalBlock[SharedIdx+4*ib+3] = tex2D( tex,
                (float) (u+4*ib-Radius+3), (float) (v-Radius) );
    }
    if ( threadIdx.y < Radius*2 ) {
        //
        // copy trailing Radius*2 rows of pixels into shared
        //
        // bis v-Radius wurde alles kopiert, fehlt also noch 2*Radius
        // also alle Werte nochmal gleich nur SharedIdx anpassen auf die richte Stelle
        // und bei der y KO einfach blockDim.y uebrspringen weil die ja schon kopiert sind
        // v-Radius .. v lief ja bis blockDim.y-1 (-Radius)
        SharedIdx = (blockDim.y+threadIdx.y) * SharedPitch;
        for ( ib = threadIdx.x; ib < BlockWidth+2*Radius; ib += blockDim.x ) {
            LocalBlock[SharedIdx+4*ib+0] = tex2D( tex,
                    (float) (u+4*ib-Radius+0), (float) (v+blockDim.y-Radius) );
            LocalBlock[SharedIdx+4*ib+1] = tex2D( tex,
                    (float) (u+4*ib-Radius+1), (float) (v+blockDim.y-Radius) );
            LocalBlock[SharedIdx+4*ib+2] = tex2D( tex,
                    (float) (u+4*ib-Radius+2), (float) (v+blockDim.y-Radius) );
            LocalBlock[SharedIdx+4*ib+3] = tex2D( tex,
                    (float) (u+4*ib-Radius+3), (float) (v+blockDim.y-Radius) );
        }
    }

    __syncthreads();

    u >>= 2;    // index as uchar4 from here
    uchar4 *pSobel = (uchar4 *) (((char *) pSobelOriginal)+v*SobelPitch);
    SharedIdx = threadIdx.y * SharedPitch;

    for ( ib = threadIdx.x; ib < BlockWidth; ib += blockDim.x ) {

        unsigned char pix00 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+0];
        unsigned char pix01 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+1];
        unsigned char pix02 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+2];
        unsigned char pix10 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+0];
        unsigned char pix11 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+1];
        unsigned char pix12 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+2];
        unsigned char pix20 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+0];
        unsigned char pix21 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+1];
        unsigned char pix22 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+2];

        uchar4 out;

        out.x = ComputeSobel(pix00, pix01, pix02,
                pix10, pix11, pix12,
                pix20, pix21, pix22, fScale );

        pix00 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+3];
        pix10 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+3];
        pix20 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+3];
        out.y = ComputeSobel(pix01, pix02, pix00,
                pix11, pix12, pix10,
                pix21, pix22, pix20, fScale );

        pix01 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+4];
        pix11 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+4];
        pix21 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+4];
        out.z = ComputeSobel( pix02, pix00, pix01,
                pix12, pix10, pix11,
                pix22, pix20, pix21, fScale );

        pix02 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+5];
        pix12 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+5];
        pix22 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+5];
        out.w = ComputeSobel( pix00, pix01, pix02,
                pix10, pix11, pix12,
                pix20, pix21, pix22, fScale );
        if ( u+ib < w/4 && v < h ) {
            pSobel[u+ib] = out;
        }
    }

    __syncthreads();
}