#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

// Texture reference for reading image
texture<guchar, 2> tex;

// arrays are optimized for 2D access so we'll use arrays
// insted of single row pointer memory addresses
hipArray *array = NULL;

// Shared Mem on the dev is declared with __shared__
extern __shared__ unsigned char LocalBlock[];


double fScale = 1.0;


void filter(
		guchar *d_image, gint width, gint height,
		guint channels, guchar *d_image_temp) {

	int add = 0;
	guint step = channels * width;

	if ( width % numThreads || height % numThreads) {
		add = 1;
	}

	switch ( filterParm.cuda_filter) {
		case GREY:
			grey<<< height, 384, 0 >>>( d_image, width, height, channels, step, filterParm);
			break;

		case BOXBIN: {
			d_boxfilter_x_tex<<< height / numThreads +add, numThreads >>>( d_image_temp, width, height, filterParm.radius);
			d_boxfilter_y_global<<< width / numThreads +add, numThreads >>>( d_image_temp, d_image, width, height, filterParm.radius, filterParm.offset, TRUE);
		}
			break;

		case SOBEL:
			sobelTex<<< height, 384, 0 >>>( d_image, width, height, channels, step, filterParm);
			break;

		case BOX: {
			d_boxfilter_x_tex<<< height / numThreads +add, numThreads >>>( d_image_temp, width, height, filterParm.radius);
			d_boxfilter_y_global<<< width / numThreads +add, numThreads >>>( d_image_temp, d_image, width, height, filterParm.radius, filterParm.offset, FALSE);
		}
			break;

		case TEST:
			test<<< height, 384, 0 >>>( d_image, width, height, channels, step, filterParm);
			break;

		case AVERAGE:
		case AVERAGEBIN: {
			if ( width % 4) {
				g_message("Width is not a multiply of 4\nThe resulting image will probably be f*ed up\n");
			}

			dim3 threads(16,16);
			int BlockWidth = 32;

			// 0==0 -> 1      0!=0 -> 0
			// Image 1280 x 1024 -> 16,4 ThreadDim and 4,256 BlockDim
			// SharedPitch -> 384
			// sharedMem -> 2304
			// SharedPitch ist durch 64Teilbar, Rechnung siehe unten bei iw&=~3
			// 16 Threads * 4 Pixel = 64
			dim3 blocks = dim3(width/(4*BlockWidth)+(0!=width%(4*BlockWidth)),
					height/threads.y+(0!=height%threads.y));
			int SharedPitch = ~0x3f&(4*(BlockWidth+2*filterParm.radius)+0x3f);
			int sharedMem = SharedPitch*(threads.y+2*filterParm.radius);

			// for the shared kernel, width must be divisible by 4
			// 3 dec = 11bin
			// ~3 dec = 00bin
			// letze beiden stellen 0 -> durch 4 teilbar
			width &= ~3;

			if ( filterParm.radius != 7 )
				printf("Wegen Optimierung ist nur Radius 7 Erlaubt in diesem Modus: Das zu sehende Bidl wird DEFINITIV Fehler enthalten\n");

			printf("AVERAGE_FAST: radius: %d  offset: %d  threads: %d,%d  blocks: %d,%d  step: %d  sharedMem: %d  width: %d\n",
					filterParm.radius, filterParm.offset, threads.x, threads.y, blocks.x, blocks.y, SharedPitch, sharedMem, width);

			AVGShared<<<blocks, threads, sharedMem>>>((uchar4 *) d_image,
					width,
					BlockWidth, SharedPitch,
					width, height, fScale,
					filterParm.radius, filterParm.offset, ( filterParm.cuda_filter == AVERAGEBIN) );
		}

			break;

		default:
			g_printerr("Filter not found");
			break;
	}

}

extern "C" void cuda_init( ) {
}

extern "C" void setupTexture( gint width, gint height) {
	hipChannelFormatDesc desc;

// 	desc = hipCreateChannelDesc<unsigned char>();
	int e = (int)sizeof( guchar) * 8;
	desc = hipCreateChannelDesc(e, 0, 0, 0, hipChannelFormatKindUnsigned);

	cutilSafeCall(hipMallocArray( &array, &desc, width, height));
}

extern "C" void bindTexture( ) {
	/* clamp x and y axis to the boarder */
	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.filterMode = hipFilterModePoint;
	tex.normalized = false;

	cutilSafeCall( hipBindTextureToArray( tex, array));
}

extern "C" void updateTexture( gint width, gint height, guchar *data, gint channels) {
	cutilSafeCall(hipMemcpyToArray(
			array,
			0, 0, /* 0 0 <- h und w offset */
			data,
			channels * sizeof( guchar) * width * height, hipMemcpyHostToDevice));
}

extern "C" void unbindTexture( ) {
	cutilSafeCall( hipUnbindTexture( tex));
}

extern "C" void deleteTexture( ) {
	cutilSafeCall( hipFreeArray( array));
}


/*
 * ALL the CUDA Functions
 */


__device__ unsigned char ComputeSobel(
		unsigned char ul, // upper left
		unsigned char um, // upper middle
		unsigned char ur, // upper right
		unsigned char ml, // middle left
		unsigned char mm, // middle (unused)
		unsigned char mr, // middle right
		unsigned char ll, // lower left
		unsigned char lm, // lower middle
		unsigned char lr, // lower right
		float fScale )
{
    short Horz = ul + 2*ml + ll - ur - 2*mr - lr;
    short Vert = ul + 2*um + ur - ll - 2*lm - lr;

    short Sum = (short) (fScale*(abs(Horz)+abs(Vert)));
    if ( Sum < 0 ) return 0; else if ( Sum > 0xff ) return 0xff;
    return (unsigned char) Sum;
}

__global__ void
sobelTex( guchar *d_image, gint width, gint height, guint channels, guint step, FilterParameter filterParm) {

	double fScale = 1.0;

	unsigned char *pSobel =
			(unsigned char *) (((char *) d_image)+blockIdx.x*step);
	for ( int i = threadIdx.x; i < width; i += blockDim.x ) {
		unsigned char pix00 = tex2D( tex, (float) i-1, (float) blockIdx.x-1 );
		unsigned char pix01 = tex2D( tex, (float) i+0, (float) blockIdx.x-1 );
		unsigned char pix02 = tex2D( tex, (float) i+1, (float) blockIdx.x-1 );
		unsigned char pix10 = tex2D( tex, (float) i-1, (float) blockIdx.x+0 );
		unsigned char pix11 = tex2D( tex, (float) i+0, (float) blockIdx.x+0 );
		unsigned char pix12 = tex2D( tex, (float) i+1, (float) blockIdx.x+0 );
		unsigned char pix20 = tex2D( tex, (float) i-1, (float) blockIdx.x+1 );
		unsigned char pix21 = tex2D( tex, (float) i+0, (float) blockIdx.x+1 );
		unsigned char pix22 = tex2D( tex, (float) i+1, (float) blockIdx.x+1 );
		pSobel[i] = ComputeSobel(
				pix00, pix01, pix02,
				pix10, pix11, pix12,
				pix20, pix21, pix22,
				fScale );
	}

}

__global__ void
box( guchar *d_image, gint width, gint height, guint channels, guint step, FilterParameter filterParm) {

	// blockIdx.x * Pitch (image.width) = Startpointer auf die Idx.x te Zeile
	unsigned char *p =
			(unsigned char *) (((char *) d_image)+blockIdx.x*step);
	int b = 0;

	for ( int i = threadIdx.x; i < width; i += blockDim.x ) {
		if(b==1) {
			if(blockIdx.x % 2 ) {
				p[i] = 255;
			} else {
				p[i] = 0;
			}
		} else {
			if(blockIdx.x % 2 ) {
				p[i] = 0;
			} else {
				p[i] = 255;
			}
		}
		b=1;
	}

}

__global__ void
test( guchar *d_image, gint width, gint height, guint channels, guint step, FilterParameter filterParm) {

	// blockIdx.x * Pitch (image.width) = Startpointer auf die Idx.x te Zeile
	unsigned char *p =
			(unsigned char *) (((char *) d_image)+blockIdx.x*step);
	int b = 0;

	for ( int i = threadIdx.x; i < width; i += blockDim.x ) {
		if(b==1) {
			if(blockIdx.x % 2 ) {
				p[i] = 255;
			} else {
				p[i] = 0;
			}
		} else {
			if(blockIdx.x % 2 ) {
				p[i] = 0;
			} else {
				p[i] = 255;
			}
		}
		b=1;
	}

}

/* OLD OUTDATED CODE */
// __global__ void greyRGB( guchar* d_image, gint width, gint height, guint channels, guint step, FilterParameter filterParm) {
// 
// 	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
// 	x *= channels;
// 	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
// 	d_image[y*step+x]   = 128;
// 	d_image[y*step+x+1] = 128;
// 	d_image[y*step+x+2] = 128;
// 
// }

__global__ void
grey( guchar* d_image, gint width, gint height, guint channels, guint step, FilterParameter filterParm) {

	for ( int i = threadIdx.x; i < width; i += blockDim.x ) {
		d_image[blockIdx.x*step+i] = filterParm.radius;
	}

}

__global__ void
sobelSharedTex(
		uchar4 *pSobelOriginal, unsigned short SobelPitch,
		short BlockWidth, short SharedPitch,
		short w, short h, float fScale, FilterParameter filterParm ) {


	int Radius = filterParm.radius;
	// pSobelOriginal > Pointer auf den Speicher in der Graka
	// SobelPitch > 1280 (BilderBreite)
	// BlockWidth > 80
	// SharedPitch > 384 // Zeilenlaenge
	// w > 1280
	// h > 1024
	// sharedMem > 2304 >> 48 * 48 = 2304
	// threads > 16,4
	// block   > 4,256
	// Radius 1 = Ich brauche links/rechts/oben/unten 1 extra pixel
	// radius darf max die haelfte des Blockes sein in x und y

	// u und v sind die KOs des Pixels, das ich kopieren will
	// u ist 4*80 = 320 -> 4*320 = 1280  -->  Der 320er Anfang jedes Blockes
	// auf u (anfang des 320er Blockes) muss dann noch der Zu nehmende Pixel addiert werden
	short u = 4*blockIdx.x*BlockWidth;
	short v = blockIdx.y*blockDim.y + threadIdx.y;
	short ib;

	// SharedIdx > Zeilenanfang vom SharedMem
	// 384 > Zeilenbreite vom SharedMem
	int SharedIdx = threadIdx.y * SharedPitch;

	// ib geht komplett durch von 0-81
	// ib geht 16er schritte
	// 4*ib = 64
	// damit hat man einheitliches lesen
	// t0 liest 4byte
	// t1 liest 4byte
	// -> 16Threads a 4byte = 64byte
	for ( ib = threadIdx.x; ib < BlockWidth+2*Radius; ib += blockDim.x ) {
		LocalBlock[SharedIdx+4*ib+0] = tex2D( tex,
				(float) (u+4*ib-Radius+0), (float) (v-Radius) );
		LocalBlock[SharedIdx+4*ib+1] = tex2D( tex,
				(float) (u+4*ib-Radius+1), (float) (v-Radius) );
		LocalBlock[SharedIdx+4*ib+2] = tex2D( tex,
				(float) (u+4*ib-Radius+2), (float) (v-Radius) );
		LocalBlock[SharedIdx+4*ib+3] = tex2D( tex,
				(float) (u+4*ib-Radius+3), (float) (v-Radius) );
	}
	if ( threadIdx.y < Radius*2 ) {
		//
		// copy trailing Radius*2 rows of pixels into shared
		//
		// bis v-Radius wurde alles kopiert, fehlt also noch 2*Radius
		// also alle Werte nochmal gleich nur SharedIdx anpassen auf die richte Stelle
		// und bei der y KO einfach blockDim.y uebrspringen weil die ja schon kopiert sind
		// v-Radius .. v lief ja bis blockDim.y-1 (-Radius)
		SharedIdx = (blockDim.y+threadIdx.y) * SharedPitch;
		for ( ib = threadIdx.x; ib < BlockWidth+2*Radius; ib += blockDim.x ) {
			LocalBlock[SharedIdx+4*ib+0] = tex2D( tex,
					(float) (u+4*ib-Radius+0), (float) (v+blockDim.y-Radius) );
			LocalBlock[SharedIdx+4*ib+1] = tex2D( tex,
					(float) (u+4*ib-Radius+1), (float) (v+blockDim.y-Radius) );
			LocalBlock[SharedIdx+4*ib+2] = tex2D( tex,
					(float) (u+4*ib-Radius+2), (float) (v+blockDim.y-Radius) );
			LocalBlock[SharedIdx+4*ib+3] = tex2D( tex,
					(float) (u+4*ib-Radius+3), (float) (v+blockDim.y-Radius) );
		}
	}

	__syncthreads();

	u >>= 2;    // index as uchar4 from here
	uchar4 *pSobel = (uchar4 *) (((char *) pSobelOriginal)+v*SobelPitch);
	SharedIdx = threadIdx.y * SharedPitch;

	for ( ib = threadIdx.x; ib < BlockWidth; ib += blockDim.x ) {

		unsigned char pix00 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+0];
		unsigned char pix01 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+1];
		unsigned char pix02 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+2];
		unsigned char pix10 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+0];
		unsigned char pix11 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+1];
		unsigned char pix12 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+2];
		unsigned char pix20 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+0];
		unsigned char pix21 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+1];
		unsigned char pix22 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+2];

		uchar4 out;

		out.x = ComputeSobel(pix00, pix01, pix02,
				pix10, pix11, pix12,
				pix20, pix21, pix22, fScale );

		pix00 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+3];
		pix10 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+3];
		pix20 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+3];
		out.y = ComputeSobel(pix01, pix02, pix00,
				pix11, pix12, pix10,
				pix21, pix22, pix20, fScale );

		pix01 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+4];
		pix11 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+4];
		pix21 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+4];
		out.z = ComputeSobel( pix02, pix00, pix01,
				pix12, pix10, pix11,
				pix22, pix20, pix21, fScale );

		pix02 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+5];
		pix12 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+5];
		pix22 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+5];
		out.w = ComputeSobel( pix00, pix01, pix02,
				pix10, pix11, pix12,
				pix20, pix21, pix22, fScale );
		if ( u+ib < w/4 && v < h ) {
			pSobel[u+ib] = out;
		}
	}

	__syncthreads();
}

// texture version
// texture fetches automatically clamp to edge of image
__global__ void
d_boxfilter_x_tex( guchar *od, int w, int h, int r) {
	float scale = 1.0f / (2*r+1);
	unsigned int y = blockIdx.x*blockDim.x + threadIdx.x;

	// In der Reihe den Vordersten Pixel berechnen
	float t = 0.0f;
	for(int x=-r; x<=r; x++) {
		t += tex2D(tex, x, y);
	}
	od[y*w] = t * scale;

	// Optimiert
	// und nun immer vorne ein Pixel abziehen
	// und hinten einen Pixel adden
	for(int x=1; x<w; x++) {
		t += tex2D(tex, x + r, y);
		t -= tex2D(tex, x - r - 1, y);
		od[y*w+x] = t * scale;
	}
}

__global__ void
d_boxfilter_y_global(guchar *id, guchar *od, int w, int h, int r, int offset, gboolean do_bin) {
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	if ( do_bin)
		d_boxfilter_y_bin(&id[x], &od[x], w, h, r, x, offset);
	else
		d_boxfilter_y(&id[x], &od[x], w, h, r, x, offset);
}

// process column
__device__ void
d_boxfilter_y(guchar *id, guchar *od, int w, int h, int r, uint x, int offset) {
	float scale = 1.0f / (2*r+1);

	float t;
	// do left edge
	t = id[0] * (r+1);
	for (int y = 1; y <= r; y++) {
		t += id[y*w];
	}
	// Average Filter
	od[0] = t * scale;

	for(int y = 1; y <= r; y++) {
		t += id[(y+r)*w];
		t -= id[0];
		// Average Filter
		od[y*w] = t * scale;
	}

	// main loop
	for(int y = r+1; y < h-r; y++) {
		t += id[(y+r)*w];
		t -= id[((y-r)*w)-w];
		od[y*w] = t * scale;
	}

	// do right edge
	for (int y = h-r; y < h; y++) {
		t += id[(h-1)*w];
		t -= id[((y-r)*w)-w];
		od[y*w] = t * scale;
	}
}

// process column
__device__ void
d_boxfilter_y_bin(guchar *id, guchar *od, int w, int h, int r, uint x, int offset) {
	float scale = 1.0f / (2*r+1);

	float t;
	// do left edge
	t = id[0] * (r+1);
	for (int y = 1; y <= r; y++) {
		t += id[y*w];
	}
	// Average Filter
	//od[0] = t * scale;
	// Binaerisierung
	if ( tex2D(tex, x, 0) < ((t * scale) + offset ))
		od[0] = 0;
	else
		od[0] = 255;

	for(int y = 1; y <= r; y++) {
		t += id[(y+r)*w];
		t -= id[0];
		// Average Filter
		//od[y*w] = t * scale;
		// Binaerisierung
		if ( tex2D(tex, x, y) < ((t * scale) + offset ))
			od[y*w] = 0;
		else
			od[y*w] = 255;
	}

	// main loop
	for(int y = r+1; y < h-r; y++) {
		t += id[(y+r)*w];
		t -= id[((y-r)*w)-w];
		//od[y*w] = t * scale;
		if ( tex2D(tex, x, y) < ((t * scale) + offset ))
			od[y*w] = 0;
		else
			od[y*w] = 255;
	}

	// do right edge
	for (int y = h-r; y < h; y++) {
		t += id[(h-1)*w];
		t -= id[((y-r)*w)-w];
		//od[y*w] = t * scale;
		if ( tex2D(tex, x, y) < ((t * scale) + offset ))
			od[y*w] = 0;
		else
			od[y*w] = 255;
	}
}


/* Fast Average Filter for a fixed radius of 7!
 * Reused Code
 */
__global__ void
AVGShared( uchar4 *pc, unsigned short step,
		short BlockWidth, short SharedPitch,
		short w, short h, float fScale,
		int radius, int offset, gboolean do_bin) {
	// pc > Pointer auf den Speicher in der Graka
	// step > 1280 (BilderBreite)
	// BlockWidth > 80
	// SharedPitch > 384
	// w > 1280
	// h > 1024
	// sharedMem > 2304 >> 48 * 48 = 2304
	// threads > 16,4
	// block   > 4,256
	// Radius 1 = Ich brauche links/rechts/oben/unten 1 extra pixel
	// radius darf max die haelfte des Blockes sein in x und y

	// u und v sind die KOs des Pixels, das ich kopieren will
	// u ist 4*80 = 320 -> 4*320 = 1280  -->  Der 320er Anfang jedes Blockes
	// auf u (anfang des 320er Blockes) muss dann noch der Zu nehmende Pixel addiert werden
	short u = 4*blockIdx.x*BlockWidth;
	short v = blockIdx.y*blockDim.y + threadIdx.y;
	short ib;

	// SharedIdx > Zeilenanfang vom SharedMem
	// 384 > Zeilenbreite vom SharedMem
	int SharedIdx = threadIdx.y * SharedPitch;


	// ib geht komplett durch von 0-81
	// ib geht 16er schritte
	// 4*ib = 64
	// damit hat man einheitliches lesen
	// t0 liest 4byte
	// t1 liest 4byte
	// -> 16Threads a 4byte = 64byte
	for ( ib = threadIdx.x; ib < BlockWidth+2*radius; ib += blockDim.x ) {
		LocalBlock[SharedIdx+4*ib+0] = tex2D( tex,
				(float) (u+4*ib-radius+0), (float) (v-radius) );
		LocalBlock[SharedIdx+4*ib+1] = tex2D( tex,
				(float) (u+4*ib-radius+1), (float) (v-radius) );
		LocalBlock[SharedIdx+4*ib+2] = tex2D( tex,
				(float) (u+4*ib-radius+2), (float) (v-radius) );
		LocalBlock[SharedIdx+4*ib+3] = tex2D( tex,
				(float) (u+4*ib-radius+3), (float) (v-radius) );
	}
	if ( threadIdx.y < radius*2 ) {
		//
		// copy trailing Radius*2 rows of pixels into shared
		//
		// bis v-Radius wurde alles kopiert, fehlt also noch 2*Radius
		// also alle Werte nochmal gleich nur SharedIdx anpassen auf die richte Stelle
		// und bei der y KO einfach blockDim.y uebrspringen weil die ja schon kopiert sind
		// v-Radius .. v lief ja bis blockDim.y-1 (-Radius)
		SharedIdx = (blockDim.y+threadIdx.y) * SharedPitch;
		for ( ib = threadIdx.x; ib < BlockWidth+2*radius; ib += blockDim.x ) {
			LocalBlock[SharedIdx+4*ib+0] = tex2D( tex,
					(float) (u+4*ib-radius+0), (float) (v+blockDim.y-radius) );
			LocalBlock[SharedIdx+4*ib+1] = tex2D( tex,
					(float) (u+4*ib-radius+1), (float) (v+blockDim.y-radius) );
			LocalBlock[SharedIdx+4*ib+2] = tex2D( tex,
					(float) (u+4*ib-radius+2), (float) (v+blockDim.y-radius) );
			LocalBlock[SharedIdx+4*ib+3] = tex2D( tex,
					(float) (u+4*ib-radius+3), (float) (v+blockDim.y-radius) );
		}
	}


	__syncthreads();


	u >>= 2;    // index as uchar4 from here
	uchar4 *pSobel = (uchar4 *) (((char *) pc)+v*step);
	SharedIdx = threadIdx.y * SharedPitch;


	unsigned char p;
	int j,k;




	for ( ib = threadIdx.x; ib < BlockWidth; ib += blockDim.x ) {

		uchar4 out;
		out.x = 10;
		out.y = 80;
		out.z = 160;
		out.w = 254;

		volatile int avg = 0;
		volatile int avg1 = 0;
		volatile int l0 = 0;
		volatile int l1 = 0;
		volatile int l2 = 0;
		volatile int ll0 = 0;
		volatile int ll1 = 0;
		volatile int ll2 = 0;

		/* prob. the fasted, but not working ):
		volatile int pixrm1 = 0;
		volatile int pixrm2 = 0;
		volatile int pixrm3 = 0;
		volatile int avg = 0;
		volatile int avg15 = 0;
		volatile int avg16 = 0;
		volatile int avg17 = 0;

		for ( j = 0; j < 15; j++) {
			for ( k = 0; k < 15; k++) {
				/*
				if ( k == 15 ) {
					avg15 += LocalBlock[SharedIdx+4*ib+j*SharedPitch+k];
					continue;
				} else if ( k == 16 ) {
					avg16 += LocalBlock[SharedIdx+4*ib+j*SharedPitch+k];
					continue;
				} else if ( k == 17 ) {
					avg17 += LocalBlock[SharedIdx+4*ib+j*SharedPitch+k];
					continue;
				}
				*/ /*
				avg1 += LocalBlock[SharedIdx+4*ib+j*SharedPitch+k];
				if ( k == 0 ) {
					pixrm1 += LocalBlock[SharedIdx+4*ib+j*SharedPitch+k];
				} else if ( k == 1 ) {
					pixrm2 += LocalBlock[SharedIdx+4*ib+j*SharedPitch+k];
				} else if ( k == 2 ) {
					pixrm3 += LocalBlock[SharedIdx+4*ib+j*SharedPitch+k];
				}
			}
		}
		p = LocalBlock[SharedIdx+4*ib+7*SharedPitch+7];
		if ( p  < ((avg / 225) + 5)) {
			out.x = 0;
		} else {
			out.x = 255;
		}

		avg -= pixrm1;

		for ( j = 0; j < 15; j++) {
			avg += LocalBlock[SharedIdx+4*ib+j*SharedPitch+15];
		}
		p = LocalBlock[SharedIdx+4*ib+7*SharedPitch+8];
		if ( p  < ((avg / 225) + 5)) {
			out.y = 0;
		} else {
			out.y = 255;
		}

		avg -= pixrm2;

		for ( j = 0; j < 15; j++) {
			avg += LocalBlock[SharedIdx+4*ib+j*SharedPitch+15];
		}
		p = LocalBlock[SharedIdx+4*ib+7*SharedPitch+9];
		if ( p  < ((avg / 225) + 5)) {
			out.z = 0;
		} else {
			out.z = 255;
		}

		avg -= pixrm3;

		for ( j = 0; j < 15; j++) {
			avg += LocalBlock[SharedIdx+4*ib+j*SharedPitch+15];
		}
		p = LocalBlock[SharedIdx+4*ib+7*SharedPitch+10];
		if ( p  < ((avg / 225) + 5)) {
			out.w = 0;
		} else {
			out.w = 255;
		}
*/ /* prob. the fasted, but not working ): */

		/* 3 bis 15 fuellen */
		avg = 0;
		avg1 = 0;
		for ( j = 0; j < 15; j++) {
			for ( k = 3; k < 15; k++) {
				avg += LocalBlock[SharedIdx+4*ib+j*SharedPitch+k];
			}
		}


		/* 0 1 2 und 15 16 17 fuellen */
		for ( j = 0; j < 15; j++) {
			l0 += LocalBlock[SharedIdx+4*ib+j*SharedPitch+0];
			l1 += LocalBlock[SharedIdx+4*ib+j*SharedPitch+1];
			l2 += LocalBlock[SharedIdx+4*ib+j*SharedPitch+2];

			ll0 += LocalBlock[SharedIdx+4*ib+j*SharedPitch+15];
			ll1 += LocalBlock[SharedIdx+4*ib+j*SharedPitch+16];
			ll2 += LocalBlock[SharedIdx+4*ib+j*SharedPitch+17];
		}

		if ( do_bin) {
			p = LocalBlock[SharedIdx+4*ib+7*SharedPitch+7];
			if ( p < (( (avg + l0 + l1 + l2) / 225 ) + offset ))
				out.x = 0;
			else
				out.x = 255;
		} else
			out.x = (avg + l0 + l1 + l2) / 225;

		if ( do_bin) {
			p = LocalBlock[SharedIdx+4*ib+7*SharedPitch+8];
			if ( p < (( (avg + l1 + l2 + ll0) / 225 ) + offset ))
				out.y = 0;
			else
				out.y = 255;
		} else
			out.y = (avg + l1 + l2 + ll0) / 225;

		if ( do_bin) {
			p = LocalBlock[SharedIdx+4*ib+7*SharedPitch+9];
			if ( p < (( (avg + l2 + ll0 + ll1) / 225 ) + offset ))
				out.z = 0;
			else
				out.z = 255;
		} else
			out.z = (avg + l2 + ll0 + ll1) / 225;

		if ( do_bin) {
			p = LocalBlock[SharedIdx+4*ib+7*SharedPitch+10];
			if ( p < (( (avg + ll0 + ll1 + ll2) / 225 ) + offset ))
				out.w = 0;
			else
				out.w = 255;
		} else
			out.w = (avg + ll0 + ll1 + ll2) / 225;


/*
		p = LocalBlock[SharedIdx+4*ib+7*SharedPitch+7];
		if ( p  < ((avg / 225) + 5)) {
			out.x = 0;
		} else {
			out.x = 255;
		}

		p = LocalBlock[SharedIdx+4*ib+7*SharedPitch+8];
		if ( p  < ((avg1 / 225) + 5)) {
			out.y = 0;
		} else {
			out.y = 255;
		}

		p = LocalBlock[SharedIdx+4*ib+7*SharedPitch+9];
		if ( p  < ((avg2 / 225) + 5)) {
			out.z = 0;
		} else {
			out.z = 255;
		}

		p = LocalBlock[SharedIdx+4*ib+7*SharedPitch+10];
		if ( p  < ((avg3 / 225) + 5)) {
			out.w = 0;
		} else {
			out.w = 255;
		}
*/

/*
		for ( j = 0; j < 15; j++) {
			for ( k = 15; k < 18; k++) {
				if ( k == 15 ) {
					avg15 += LocalBlock[SharedIdx+4*ib+j*SharedPitch+k];
					continue;
				} else if ( k == 16 ) {
					avg16 += LocalBlock[SharedIdx+4*ib+j*SharedPitch+k];
					continue;
				} else if ( k == 17 ) {
					avg17 += LocalBlock[SharedIdx+4*ib+j*SharedPitch+k];
					continue;
				}
			}
		}

		avg15 += ((avg - pixrm1) / 225);
		avg16 += ((avg15 - pixrm2) / 225);
		avg17 += ((avg16 - pixrm3) / 225);

		p = LocalBlock[SharedIdx+4*ib+7*SharedPitch+8];
		if ( p  < (avg15 + 5) ) {
			out.y = 0;
		} else {
			out.y = 255;
		}

		p = LocalBlock[SharedIdx+4*ib+7*SharedPitch+9];
		if ( p  < (avg16 + 5)) {
			out.z = 0;
		} else {
			out.z = 255;
		}

		p = LocalBlock[SharedIdx+4*ib+7*SharedPitch+10];
		if ( p  < (avg17 + 5)) {
			out.w = 0;
		} else {
			out.w = 255;
		}
*/

/*
		avg -= pixrm2;

		for ( j = 0; j < 15; j++) {
			avg += LocalBlock[SharedIdx+4*ib+j*SharedPitch+16];
		}
		p = LocalBlock[SharedIdx+4*ib+7*SharedPitch+9];
		if ( p  < ((avg / 225) + 5)) {
			out.z = 0;
		} else {
			out.z = 255;
		}

		avg -= pixrm3;

		for ( j = 0; j < 15; j++) {
			avg += LocalBlock[SharedIdx+4*ib+j*SharedPitch+17];
		}
		p = LocalBlock[SharedIdx+4*ib+7*SharedPitch+10];
		if ( p  < ((avg / 225) + 5)) {
			out.w = 0;
		} else {
			out.w = 255;
		}
*/


		if ( u+ib < w/4 && v < h ) {
#ifdef __DEVICE_EMULATION__
			printf("1: Wert x: %d  Wert y: %d  Wert z: %d  Wert w: %d\n", out.x, out.y, out.z, out.w);
#endif
			pSobel[u+ib] = out;
		}

	}

	__syncthreads();
}
